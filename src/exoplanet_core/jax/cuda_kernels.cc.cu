#include "hip/hip_runtime.h"
#include <exoplanet/kepler.h>

#include "cuda_kernels.h"
#include "kernel_helpers.h"

namespace exoplanet {

namespace {
template <typename Scalar>
__global__ void KeplerKernel(int N, const Scalar* M, const Scalar* ecc, Scalar* cosf,
                             Scalar* sinf) {
  Scalar a, b;
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    exoplanet::kepler::solve_kepler<Scalar>(M[idx], ecc[idx], &a, &b);
    cosf[idx] = a;
    sinf[idx] = b;
  }
}
}  // namespace

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

struct KeplerDescriptor {
  int N;
};

std::string BuildCudaKeplerDescriptor(int N) {
  return PackDescriptorAsString(KeplerDescriptor{N});
}

void CudaKepler(hipStream_t stream, void** buffers, const char* opaque, std::size_t opaque_len) {
  const double* M = reinterpret_cast<const double*>(buffers[0]);
  const double* ecc = reinterpret_cast<const double*>(buffers[1]);
  double* cosf = reinterpret_cast<double*>(buffers[2]);
  double* sinf = reinterpret_cast<double*>(buffers[3]);

  const auto& descriptor = *UnpackDescriptor<KeplerDescriptor>(opaque, opaque_len);
  int N = descriptor.N;

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (N + block_dim - 1) / block_dim);

  KeplerKernel<<<grid_dim, block_dim, 0, stream>>>(N, M, ecc, cosf, sinf);
  ThrowIfError(hipGetLastError());
}

}  // namespace exoplanet
