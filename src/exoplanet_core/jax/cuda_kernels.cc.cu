#include "hip/hip_runtime.h"
#include <exoplanet/exoplanet.h>

#include <limits>

#include "cuda_kernels.h"
#include "kernel_helpers.h"
namespace exoplanet {

namespace {
template <typename Scalar>
__global__ void KeplerKernel(int N, const Scalar* M, const Scalar* ecc, Scalar* sinf,
                             Scalar* cosf) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    kepler::solve_kepler<Scalar>(M[idx], ecc[idx], sinf + idx, cosf + idx);
  }
}

template <typename Scalar>
__global__ void QuadSolutionVectorKernel(Scalar eps, int N, const Scalar* b, const Scalar* r,
                                         Scalar* s, Scalar* dsdb, Scalar* dsdr) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
    int offset = 3 * idx;
    int sgn = exoplanet::sgn(b[idx]);
    limbdark::quad_solution_vector<true>(eps, std::abs(b[idx]), r[idx], s + offset, dsdb + offset,
                                         dsdr + offset);
    dsdb[offset] *= sgn;
    dsdb[offset + 1] *= sgn;
    dsdb[offset + 2] *= sgn;
  }
}

}  // namespace

void ThrowIfError(hipError_t error) {
  if (error != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(error));
  }
}

struct SizeDescriptor {
  int N;
};

std::string BuildCudaDescriptor(int N) { return PackDescriptorAsString(SizeDescriptor{N}); }

void CudaKepler(hipStream_t stream, void** buffers, const char* opaque, std::size_t opaque_len) {
  const double* M = reinterpret_cast<const double*>(buffers[0]);
  const double* ecc = reinterpret_cast<const double*>(buffers[1]);
  double* sinf = reinterpret_cast<double*>(buffers[2]);
  double* cosf = reinterpret_cast<double*>(buffers[3]);

  const auto& descriptor = *UnpackDescriptor<SizeDescriptor>(opaque, opaque_len);
  int N = descriptor.N;

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (N + block_dim - 1) / block_dim);

  KeplerKernel<<<grid_dim, block_dim, 0, stream>>>(N, M, ecc, sinf, cosf);
  ThrowIfError(hipGetLastError());
}

void CudaQuadSolutionVector(hipStream_t stream, void** buffers, const char* opaque,
                            std::size_t opaque_len) {
  const double* b = reinterpret_cast<const double*>(buffers[0]);
  const double* r = reinterpret_cast<const double*>(buffers[1]);
  double* s = reinterpret_cast<double*>(buffers[2]);
  double* dsdb = reinterpret_cast<double*>(buffers[3]);
  double* dsdr = reinterpret_cast<double*>(buffers[4]);

  const auto& descriptor = *UnpackDescriptor<SizeDescriptor>(opaque, opaque_len);
  int N = descriptor.N;

  const double eps = std::numeric_limits<double>::epsilon();

  const int block_dim = 128;
  const int grid_dim = std::min<int>(1024, (N + block_dim - 1) / block_dim);

  QuadSolutionVectorKernel<<<grid_dim, block_dim, 0, stream>>>(eps, N, b, r, s, dsdb, dsdr);
  ThrowIfError(hipGetLastError());
}

}  // namespace exoplanet
